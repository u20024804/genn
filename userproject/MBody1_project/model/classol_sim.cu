/*--------------------------------------------------------------------------
   Author: Thomas Nowotny
  
   Institute: Institute for Nonlinear Science
              University of California San Diego
              La Jolla, CA 92093-0402
  
   email to:  tnowotny@ucsd.edu
  
   initial version: 2002-09-26
  
--------------------------------------------------------------------------*/

//--------------------------------------------------------------------------
/*! \file classol_sim.cu

\brief Main entry point for the classol (CLASSification in OLfaction) model simulation. Provided as a part of the complete example of simulating the MBody1 mushroom body model. 
*/
//--------------------------------------------------------------------------


#include "classol_sim.h"

//--------------------------------------------------------------------------
/*! \brief This function is the entry point for running the simulation of the MBody1 model network.
*/
//--------------------------------------------------------------------------


int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "usage: classol_sim <basename> <CPU=0, GPU=1> \n");
    return 1;
  }
  int GPUarg= atoi(argv[2]);
  string OutDir = toString(argv[1]) +"_output";
  string name;
  name= OutDir+ "/"+ toString(argv[1]) + toString(".time");
  FILE *timef= fopen(name.c_str(),"a");  

  int which;
  if (GPUarg > 1) {
     which= 1;
     nGPU= GPUarg-2;
  }    
  else {
     which= GPUarg;	
     nGPU= AUTODEVICE;
  }
  patSetTime= (int) (PAT_TIME/DT);
  patFireTime= (int) (PATFTIME/DT);
  fprintf(stdout, "# DT %f \n", DT);
  fprintf(stdout, "# T_REPORT_TME %f \n", T_REPORT_TME);
  fprintf(stdout, "# SYN_OUT_TME %f \n",  SYN_OUT_TME);
  fprintf(stdout, "# PATFTIME %f \n", PATFTIME); 
  fprintf(stdout, "# patFireTime %d \n", patFireTime);
  fprintf(stdout, "# PAT_TIME %f \n", PAT_TIME);
  fprintf(stdout, "# patSetTime %d \n", patSetTime);
  fprintf(stdout, "# TOTAL_TME %f \n", TOTAL_TME);
  
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.Vm"); 
  FILE *osf= fopen(name.c_str(),"w");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.st"); 
  FILE *osf2= fopen(name.c_str(),"w");

#ifdef TIMING
  name= OutDir+ "/"+ toString(argv[1]) + toString(".timingprofile"); 
  FILE *timeros= fopen(name.c_str(),"w");
  sdkCreateTimer(&timer_gen);
  double tme;
#endif

  //-----------------------------------------------------------------
  // build the neuronal circuitery
  classol locust;

#ifdef TIMING
  sdkStartTimer(&timer_gen);
#endif

  fprintf(stdout, "# reading PN-KC synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".pnkc");
  FILE *f= fopen(name.c_str(),"r");
  locust.read_pnkcsyns(f);
  fclose(f);

#ifdef TIMING
  sdkStopTimer(&timer_gen);
  tme= sdkGetTimerValue(&timer_gen);
  fprintf(timeros, "%% Reading PN-KC synapses: %f \n", tme);
  sdkResetTimer(&timer_gen);
  sdkStartTimer(&timer_gen);
#endif

  fprintf(stdout, "# reading PN-LHI synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".pnlhi");
  f= fopen(name.c_str(), "r");
  locust.read_pnlhisyns(f);
  fclose(f);   

#ifdef TIMING
  sdkStopTimer(&timer_gen);
  tme= sdkGetTimerValue(&timer_gen);
  fprintf(timeros, "%% Reading PN-LHI synapses: %f \n", tme);
  sdkResetTimer(&timer_gen);
  sdkStartTimer(&timer_gen);
#endif
  
  fprintf(stdout, "# reading KC-DN synapses ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".kcdn");
  f= fopen(name.c_str(), "r");
  locust.read_kcdnsyns(f);

#ifdef TIMING
  sdkStopTimer(&timer_gen);
  tme= sdkGetTimerValue(&timer_gen);
  fprintf(timeros, "%% Reading KC-DN synapses: %f \n", tme);
  sdkResetTimer(&timer_gen);
  sdkStartTimer(&timer_gen);
#endif

  fprintf(stdout, "# reading input patterns ... \n");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".inpat");
  f= fopen(name.c_str(), "r");
  locust.read_input_patterns(f);
  fclose(f);

#ifdef TIMING
  sdkStopTimer(&timer_gen);
  tme= sdkGetTimerValue(&timer_gen);
  fprintf(timeros, "%% Reading input patterns: %f \n", tme);
  sdkResetTimer(&timer_gen);
  sdkStartTimer(&timer_gen);
#endif

  locust.generate_baserates();
  if (which == GPU) {
    locust.allocate_device_mem_patterns();
  }
  locust.init(which);         // this includes copying g's for the GPU version

#ifdef TIMING
  sdkStopTimer(&timer_gen);
  tme= sdkGetTimerValue(&timer_gen);
  fprintf(timeros, "%% Initialisation: %f \n", tme);
  sdkResetTimer(&timer_gen);
#endif

  fprintf(stdout, "# neuronal circuitery built, start computation ... \n\n");

  //------------------------------------------------------------------
  // output general parameters to output file and start the simulation

  fprintf(stdout, "# We are running with fixed time step %f \n", DT);
  fprintf(stdout, "# initial wait time execution ... \n");

  t= 0.0;
  int done= 0;
  float last_t_report=  t;
  locust.run(DT, which);
  float synwriteT= 0.0f;
  float lastsynwrite= 0.0f;
  int synwrite= 0;
  timer.startTimer();
  while (!done) 
  {
    if (which == GPU) {
      locust.getSpikeNumbersFromGPU();
      locust.getSpikesFromGPU();
    }
    locust.run(DT, which); // run next batch
    // if (which == GPU) {  
//	pullDNfromDevice();
    //   }
    
#ifdef TIMING
    if (which == CPU) {
	fprintf(timeros, "%f %f %f \n", sdkGetTimerValue(&neuron_timer), sdkGetTimerValue(&synapse_timer), sdkGetTimerValue(&learning_timer));
    }
    else {
	fprintf(timeros, "%f %f %f \n", neuron_tme, synapse_tme, learning_tme);
    }
#endif

    locust.sum_spikes();
    locust.output_spikes(osf2, which);

 /*   fprintf(osf, "%f ", t);
    //  for (int i= 0; i < 100; i++) {
    //     fprintf(osf, "%f ", VDN[i]);
    //   }
    // fprintf(osf,"\n");
*/
    // report progress
    if (t - last_t_report >= T_REPORT_TME)
    {
      fprintf(stdout, "time %f \n", t);
      last_t_report= t;
    }
    // output synapses occasionally
    if (synwrite) {
       lastsynwrite= synwriteT;
       name= OutDir+ "/"+ tS(argv[1]) + tS(".") + tS((int) synwriteT) + tS(".syn"); 
       f= fopen(name.c_str(),"w");
       locust.write_kcdnsyns(f);
       fclose(f);
       synwrite= 0;
    }
    if (t - lastsynwrite >= SYN_OUT_TME) {
       locust.get_kcdnsyns();
       synwrite= 1;
       synwriteT= t;
    }
    done= (t >= TOTAL_TME);
  }
  timer.stopTimer();
  cerr << "output files are created under the current directory." << endl;
  fprintf(timef, "%d %u %u %u %u %u %.4f %.2f %.1f %.2f\n",which, locust.model.sumNeuronN[locust.model.neuronGrpN-1], locust.sumPN, locust.sumKC, locust.sumLHI, locust.sumDN, timer.getElapsedTime(),VDN[0], TOTAL_TME, DT);
  fclose(osf);
  fclose(osf2);
  fclose(timef);
  freeDeviceMem();
  hipDeviceReset();

#ifdef TIMING
  fclose(timeros);
#endif

  return 0;
}
